
#include "hip/hip_runtime.h"

#include "hipblas.h"
#include <stdio.h>
#include <vector>

hipComplex multy(hipComplex&a, hipComplex&b)
{
	hipComplex c;
	c.x = a.x*b.x - a.y*b.y;
	c.y = a.x*b.y + a.y*b.x;
	return c;
}


int main()
{
	const int m = 1000;
	const int n = 1600;
	const int k = 1000;
	hipComplex *dm_A, *dm_B, *d_C;
	std::vector<hipComplex> C(m*n);
	std::vector<hipComplex> h_c(m*n);
	//Malloc
	hipMallocManaged((void**)&dm_A, sizeof(hipComplex)*m*k);
	hipMallocManaged((void**)&dm_B, sizeof(hipComplex)*k*n);
	hipMalloc((void**)&d_C, sizeof(hipComplex)*m*n);
	//Init data
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < k; j++)
		{
			//������
			dm_A[i*k + j].x = i*j;
			dm_A[i*k + j].y = j;
		}
	}
	for (int i = 0; i < k; i++)
	{
		for (int j = 0; j < n; j++)
		{
			//������
			dm_B[i*n + j].x = j;
			dm_B[i*n + j].y = i*j;
		}
	}
	//cublas
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//Ain Arow�������ȵ�
	hipComplex alpha, beta;
	alpha.x = 1; alpha.y = 0;
	beta.x = 0; beta.y = 0;
	hipblasCgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		n, m, k,
		&alpha,
		dm_B, n,
		dm_A, k,
		&beta,
		d_C, n);
	hipDeviceSynchronize();
	hipMemcpy(&(h_c[0]), d_C, sizeof(hipComplex)*m*n, hipMemcpyDeviceToHost);
	hipblasDestroy(handle);

	//cpu
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < n; j++)
		{
			hipComplex temp;
			temp.x = 0; temp.y = 0;
			for (int t = 0; t < k; t++)
			{
				hipComplex b = multy(dm_A[i*k + t], dm_B[t*n + j]);
				temp.x += b.x;
				temp.y += b.y;
			}
			C[i*n + j] = temp;
		}
	}
	//Verify
	system("pause");
}
